#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include "manual_vector.h"
#include "unified.h"

class State : public UnifiedMemoryClass {
public:
  GenericVector<double> x;
  int size;
  int device_fill_size;
  double xsum;
  State() {}
};

__global__ void double_tens(State* state) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid == 0) {

    for (int i = 0; i < state->x.size(); i++) {
        state->x[i] *= 2;
    }    

    for (int i = 0; i < state->x.size(); i++) {
        state->xsum += state->x[i];
    }
  }
}

int main(int argc, char* argv[]) {
  State* state = new State();
  int size = 10;
  hipError_t cuda_status = hipSuccess;

  state->size = size;
  state->xsum = 0.0;

  std::cout << "pushing data into vector on host ..." << std::endl;  
  for (int i = 0; i < size; i++) {
    state->x.push_back(10.0);
  }

  for (int i = 0; i < size; i++) {
      std::cout << "vector at " << i << " is " << state->x[i] << std::endl;
  }

  std::cout << "launching work kernel ..." << std::endl;
  double_tens<<<1,1>>>(state);

  std::cout << "cuda status is " << hipGetErrorString(cuda_status) << std::endl;  

  cuda_status = hipDeviceSynchronize();
  std::cout << "cuda status is " << hipGetErrorString(cuda_status) << std::endl;    
  assert(cuda_status == hipSuccess);

  std::cout << "sum of 2*10 " << size << " times is: " << state->xsum << std::endl;

  if (state->xsum == 2*10.0*size) std::cout << "SUCCESS!" << std::endl;
  else std::cout << "ERROR!" << std::endl;
  
  return 0;
}
